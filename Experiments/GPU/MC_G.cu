#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <math.h>
#include <stdio.h>
#include <assert.h>
#include <fstream>
#include <time.h>

void image_convolution(float * input,float* output, int img_height, int img_width, const int r, float & gpu_elapsed_time_ms)
{


    // initialize kernel here
    int kernel_height = r;
    int kernel_width = r;

    float *kernel;
    kernel = new float[r*r];

    for (int i = 0; i < r*r; i++){
        kernel[i] = rand() % 10 + 1;
    }


    float * mask = new float[kernel_height*kernel_width];
    for (int i = 0; i < kernel_height*kernel_width; i++)
    {
        mask[i] = kernel[i];
    }

    float * d_input, * d_output, * d_kernel;
    hipMalloc(&d_input, img_width*img_height*sizeof(float));
    hipMalloc(&d_output, img_width*img_height*sizeof(float));
    hipMalloc(&d_kernel, kernel_height*kernel_width*sizeof(float));

    hipMemcpy(d_input, input, img_width*img_height*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, mask, kernel_height*kernel_width*sizeof(float), hipMemcpyHostToDevice);
    dim3 blocksize(16,16);
    dim3 gridsize;
    gridsize.x=(img_width+blocksize.x-1)/blocksize.x;
    gridsize.y=(img_height+blocksize.y-1)/blocksize.y;




    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEventRecord(start, 0);

    image_convolution_kernel<<<gridsize,blocksize>>>(d_input,d_output,d_kernel,img_width,img_height,kernel_width,kernel_height);
    hipMemcpy(output, d_output, img_width*img_height*sizeof(float), hipMemcpyDeviceToHost);


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
}

int main(){
    
    // change here to customize output filename
    string FILENAME = "matrix_conv_gpu_500_points_Quadro.csv";
    // number of instances of data generated
    int NUM = 500;

    std::ofstream ofile;
    ofile.open(FILENAME);

    for (int iterator = 0; iterator < NUM; iterator++) {


        if (iterator % 10 == 0) std::cout << "iter: " << iterator << std::endl;

        float *in, *out;
        int m = rand() % 1024 + 10;
        int n = rand() % 1024 + 10;
        int is = n * m;

        int r = (rand() % 3 + 1) * 2 + 1;

        in = new float[is];
        out = new float[is];


        // density
        int power;
        double d;


        power = rand() % int((log2(double(m * n)) + 1));
        d = 1 / pow(2, power);


        if (d <= 0.5) {
            int count_a = m * n * d;
            for (int it = 0; it < count_a; it++) {
                int i = rand() % m;
                int j = rand() % n;

                in[i * n + j] = rand() % 1024 + 1;
            }
        } else {
            for (int i = 0; i < m * n; i++) {
                in[i] = rand() % 1024 + 1;
            }
        }

        float time;

        image_convolution(in, out, n, m, r, time);
        int c = (m-r+1)*(n-r+1)*r*r;
        ofile << time / 1000;
        ofile << "," << m << "," << n << "," << r << "," << d << "," << c << ",\n";


    }

    ofile.close();
    return 0;
}