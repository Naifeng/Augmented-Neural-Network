#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>
#include <fstream>
#include <time.h>
#include <thrust/device_vector.h>
#include <math.h>

#define BLOCK_SIZE 16

__global__ void MM(int *a,int *b, int *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m) {
        for(int i = 0; i < n; i++) {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 

int main(int argc, char const *argv[])
{   
    // open the output file
    std::ofstream ofile;
    // customize output filename
    ofile.open("matrix_matrix_gpu_5000_points.csv"); 
    // number of instances of data generated
    int NUM = 5000;

    for (int iterator = 0; iterator < NUM; iterator++) { 

        if (iterator % 10 == 0) std::cout << "iter: " << iterator << std::endl;

        // size
        int m, n, k;
        m = rand() % 1024 + 1;
        n = rand() % 1024 + 1;
        k = rand() % 1024 + 1;

        // density
        int power1, power2;
        double d1,d2;

        power1 = rand()%int((log2(double(m*n))+1));
        d1 = 1/pow(2,power1);

        power2 = rand()%int((log2(double(n*k))+1));
        d2 = 1/pow(2,power2);


        // [m*n] * [n*k]

        // allocate memory in host RAM
        int *h_a, *h_b, *h_c;
        hipHostMalloc((void **) &h_a, sizeof(int) * m * n);
        hipHostMalloc((void **) &h_b, sizeof(int) * n * k);
        hipHostMalloc((void **) &h_c, sizeof(int) * m * k);

        // initialize matrix A
        // if A is a sparse matrix 
        if (d1 <= 0.5){
            int count_a = m * n * d1;
            for (int it = 0; it < count_a; it++){
                // approximation
                int i = rand() % m;
                int j = rand() % n;
                h_a[i*n+j] = rand() % 1024 + 1;
            }
        }
        // if A is a dense matrix
        else{
            for (int i = 0; i < m; i++){
                for (int j = 0; j < n; j++){
                    h_a[i*n+j] = rand() % 1024 + 1;
                }
           
            }
        }

        // initialize matrix B
        // if B is a sparse matrix 
        if (d2 <= 0.5){
            int count_b = n * k * d2;
            for (int it = 0; it < count_b; it++){
                // approximation
                int i = rand() % n;
                int j = rand() % k;
                h_b[i*k+j] = rand() % 1024 + 1;
            }
        }
        // if B is a dense matrix
        else{
            for (int i = 0; i < n; i++){
                for (int j = 0; j < k; j++){
                    h_b[i*k+j] = rand() % 1024 + 1;
                }
           
            }
        }


        // Allocate memory space on the device
        int *d_a, *d_b, *d_c;
        hipMalloc((void **) &d_a, sizeof(int) * m * n);
        hipMalloc((void **) &d_b, sizeof(int) * n * k);
        hipMalloc((void **) &d_c, sizeof(int) * m * k);

        hipMemcpy(d_a, h_a, sizeof(int) * m * n, hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, sizeof(int) * n * k, hipMemcpyHostToDevice);


        float gpu_elapsed_time_ms;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
        unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
        dim3 dimGrid(grid_cols, grid_rows);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);


        hipEventRecord(start, 0);

        // launch kernel
        MM << < dimGrid, dimBlock >> > (d_a, d_b, d_c, m, n, k);

        hipMemcpy(h_c, d_c, sizeof(int) * m * k, hipMemcpyDeviceToHost);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);


        hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);

        int c = m*n*k;
        ofile << gpu_elapsed_time_ms/1000;
        ofile << "," << m << "," << n << "," << k << ",";
        ofile << d1 << "," << d2 << "," << c << ",\n";


        // free memory
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        hipHostFree(h_a);
        hipHostFree(h_b);
        hipHostFree(h_c);

    }

    ofile.close();
    return 0;
}